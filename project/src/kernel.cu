#include "hip/hip_runtime.h"
﻿#include "kernel.cuh"
#include "calculate.h"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>

__global__ void addKernel(int* c, const int* a, const int* b, int size)
{
	int i = threadIdx.x;
	if (i < size)
		c[i] = a[i] + b[i];
}

std::vector<int> calculate::addGPU(std::vector<int> numb1, std::vector<int> numb2) {
	std::vector<int> summ; 
	int size=0;

	thrust::device_vector<int> d_numb1(numb1.size(), 0);
	thrust::device_vector<int> d_numb2(numb2.size(), 0);

	(numb1.size() > numb2.size()) ? size = numb2.size() : size = numb1.size();

	thrust::device_vector<int> d_summ(size, 0);

	thrust::copy(numb1.begin(), numb1.end(), d_numb1.begin());
	thrust::copy(numb2.begin(), numb2.end(), d_numb2.begin());

	//int* d_raw1 = thrust::raw_pointer_cast(&d_numb1[0]);
	int* d_raw1 = thrust::raw_pointer_cast(d_numb1.data());
	int* d_raw2 = thrust::raw_pointer_cast(d_numb2.data());

	int* d_raw_s = thrust::raw_pointer_cast(d_summ.data());
	
	// Launch a kernel on the GPU with one thread for each element.
	addKernel <<<1, size >>> (d_raw_s, d_raw1, d_raw2, size);
	// void* args[] = { &d_raw_s, &d_raw1, &d_raw2, &size };
	// hipLaunchKernel<void>(&addKernel, 1, d_numb1.size(), args);

	thrust::host_vector<int> result = d_summ;

	//for (const int item : result)
	//	std::cout << item << " ";
	//std::cout << std::endl;

	summ.resize(d_summ.size());
	thrust::copy(d_summ.begin(), d_summ.end(), summ.begin());

	return summ;
}